
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h"
#define IDX2C(i,j,ld) (((j)*(ld))+(i))
#define n 6
#define k 1
int main(void){
  hipblasHandle_t handle;
  int i,j;
  float* a;
  float* x;
  float* y;
  hipMallocManaged(&a,n*n*sizeof(float));
  hipMallocManaged(&x,n*sizeof(float));
  hipMallocManaged(&y,n*sizeof(float));
  int ind=11;
  for(i = 0; i < n; i++) a[i*n] = (float)ind++;
  for(i = 0; i < n-1; i++) a[i*n+1] = (float)ind++;
  for(i = 0; i < n; i++){x[i]=1.0f; y[i]=0.0f;};

  hipblasCreate(&handle);
  float al = 1.0f;
  float bet = 1.0f;
  hipblasSsbmv(handle, HIPBLAS_FILL_MODE_LOWER,n,k,&al,a,n,x,1,&bet,y,1);
  hipDeviceSynchronize();
  printf("y after ssbmv:\n");
  for(j = 0; j < n; j++){
    printf("%7.0f\n",y[j]);
  }
  
  hipFree(a);
  hipFree(x);
  hipFree(y);
  hipblasDestroy(handle);
  return EXIT_SUCCESS;
}
