
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h"
#define IDX2C(i,j,ld) (((j)*(ld))+(i))  // 将一行数组转化成一个矩阵 先纵向一列，j为列数，i为行数
#define m 6
#define n 5
int main(void){
  hipblasHandle_t handle;
  int i, j;
  float* a;
  float* x;
  float* y;
  hipMallocManaged(&a,m*n*sizeof(float));
  hipMallocManaged(&x,n*sizeof(float));
  hipMallocManaged(&y,n*sizeof(float));
  int ind=11;
  for(j = 0; j < n; j++){
    for(i = 0; i < m; i++){
      a[IDX2C(i,j,m)] = (float)ind++;
    }
  }
  for(i = 0; i < n; i++) x[i]=1.0f;
  for(i = 0; i < m; i++) y[i]=0.0f;

  // device
  hipblasCreate(&handle);
  float al = 1.0f;
  float bet = 0.0f;
  hipblasSgemv(handle, HIPBLAS_OP_N, m, n, &al, a, m, x, 1, &bet, y, 1);
  hipDeviceSynchronize();
  printf("y after sgemv:\n");
  for(j = 0; j < m; j++){
    printf("%7.0f\n", y[j]);
    printf("\n");
  }
  hipFree(a);
  hipFree(x);
  hipFree(y);
  hipblasDestroy(handle);
  return EXIT_SUCCESS;
}
