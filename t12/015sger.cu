
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h"
#define IDX2C(i,j,ld) (((j)*(ld))+(i))
#define m 6
#define n 5
int main(void){
  hipblasHandle_t handle;
  int i,j;
  float* a;
  float* x;
  float* y;
  hipMallocManaged(&a,m*n*sizeof(float));
  hipMallocManaged(&x,m*sizeof(float));
  hipMallocManaged(&y,n*sizeof(float));
  
  int ind = 11;
  float al = 2.0f;
  for(j = 0; j < n; j++){ 
    for(i = 0; i < m; i++){
      a[IDX2C(i,j,m)] = (float)ind++;
    }
  }
  for(i = 0; i < m; i++) x[i] = 1.0f;
  for(i = 0; i < n; i++) y[i] = 1.0f;

  hipblasCreate(&handle);
  hipblasSger(handle,m,n,&al,x,1,y,1,a,m);
  hipDeviceSynchronize();
  
  printf("a after sger:\n");
  for(int i = 0; i < m; i++){
    for(int j = 0; j < n; j++){
      printf("%7.0f", a[IDX2C(i,j,m)]);
    }
    printf("\n");
  }
  hipFree(a);
  hipFree(x);
  hipFree(y);
  hipblasDestroy(handle);
  return EXIT_SUCCESS;
}
