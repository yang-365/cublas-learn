
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h"
#define n 6
int main(void){
  hipblasHandle_t handle;
  int j;
  float* x;
  hipMallocManaged((void**)&x, n*sizeof(float));
  for(j  = 0; j < n; j++){
    x[j] = (float)j;
  }

  // device
  float result;
  hipblasCreate(&handle);
  hipblasSnrm2(handle, n, x, 1, &result);
  hipDeviceSynchronize();
  printf("Euclidean norm: %7.3f\n", result);
  hipFree(x);
  hipblasDestroy(handle);
  return EXIT_SUCCESS;
}
