
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h"
#define n 6
int main(void){
  hipblasHandle_t handle;  // CUBLAS context
  int j;
  float* x;
  float* y;
  hipMallocManaged((void**)&x, n*sizeof(float));
  for(j = 0; j < n; j++){
    x[j] = (float)j;
  }  
  hipMallocManaged((void**)&y, n*sizeof(float));
  
  // device 
  hipblasCreate(&handle);
  hipblasScopy(handle, n ,x, 1, y, 1);
  hipDeviceSynchronize();
  printf("y after scopy:");
  for(j = 0; j < n; j++){
    printf("%4.0f ", y[j]);
  }
  printf("\n");

  hipFree(x);
  hipFree(y);
  hipblasDestroy(handle);
  return EXIT_SUCCESS;

}
