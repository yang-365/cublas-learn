
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h"
#define IDX2C(i,j,ld) (((j)*(ld))+(i))
#define m 5
#define n 6
#define ku 2
#define kl 1
int main(void){
  hipblasHandle_t handle;
  int i,j;
  float* a;
  float* x;
  float* y;
  hipMallocManaged(&a,m*n*sizeof(float));
  hipMallocManaged(&x,n*sizeof(float));
  hipMallocManaged(&y,n*sizeof(float));
  int ind=11;
  for(i = ku; i < n; i++) a[IDX2C(0,i,m)]=(float)ind++;
  for(i = ku-1; i < n; i++) a[IDX2C(1,i,m)]=(float)ind++;
  for(i = 0; i < n-1; i++) a[IDX2C(ku,i,m)]=(float)ind++;
  for(i = 0; i < n-2; i++) a[IDX2C(ku+1,i,m)]=(float)ind++;
  
  for(i = 0; i < n; i++) x[i] = 1.0f;
  for(i = 0; i < m; i++) y[i] = 0.0f;
  
  hipblasCreate(&handle);
  float al = 1.0f;
  float bet = 1.0f;
  hipblasSgbmv(handle, HIPBLAS_OP_N, m, n, kl, ku, &al, a, m, x, 1, &bet, y, 1);
  hipDeviceSynchronize();
  
  printf("y after Sgbmv:\n");
  for(j = 0; j < m; j++){
      printf("%7.0f", y[j]);
      printf("\n");
  }
  hipFree(a);
  hipFree(x);
  hipFree(y);
  hipblasDestroy(handle);
  return EXIT_SUCCESS;
}
