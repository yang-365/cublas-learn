
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h"
#define n 6
int main(void){
  hipblasHandle_t handle;
  int j;
  float* x;
  float* y;
  hipMallocManaged(&x, n*sizeof(float));
  hipMallocManaged(&y, n*sizeof(float));
  for(j = 0; j < n; j++){
    x[j] = (float)j;
    y[j] = (float)j*j;
  }

  // device
  hipblasCreate(&handle);
  hipblasSswap(handle, n, x, 1, y, 1);
  hipDeviceSynchronize();
  printf("x after sswap:");
  for(j = 0; j < n; j++){
    printf("%7.2f ", x[j]);
  }
  printf("\n");

  printf("y after sswap:");
  for(j = 0; j < n; j++){
    printf("%7.2f ", y[j]);
  }
  printf("\n");
  hipFree(x);
  hipFree(y);
  hipblasDestroy(handle);
  return EXIT_SUCCESS;

}
