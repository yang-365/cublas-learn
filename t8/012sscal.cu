
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h"
#define n 6
int main(void){
  hipblasHandle_t handle;
  int j;
  float *x;
  hipMallocManaged((void**)&x, n*sizeof(float));
  for(j = 0; j < n; j++){
    x[j] = (float)j;
  }

  hipblasCreate(&handle);
  float al = 2.0;
  hipblasSscal(handle, n, &al, x, 1);
  hipDeviceSynchronize();
  printf("x after sscal:");
  for(j = 0; j < n; j++){
    printf("%7.3f ", x[j]);
  }
  printf("\n");

  hipFree(x);
  hipblasDestroy(handle);
  return EXIT_SUCCESS;
}
