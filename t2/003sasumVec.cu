
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h"
#define n 6
int main(void){
  hipblasHandle_t handle;
  int j;
  float *x;
  hipMallocManaged(&x, n*sizeof(float));
  for(j = 0; j < n; j++){
    x[j] = (float)j;
  }
  printf("x: ");
  for(j = 0; j < n; j++){
    printf("%4.0f ", x[j]);
  }
  printf("\n");
  // DEVICE
  hipblasCreate(&handle);  // initialize CUBLAS context
  float result;
  // add sbsolute value 
  hipblasSasum(handle, n, x, 1, &result);
  hipDeviceSynchronize();
  printf("sum of absolute value %4.0f\n", result);
  
  hipFree(x);
  hipblasDestroy(handle);
  return EXIT_SUCCESS;
  
}
