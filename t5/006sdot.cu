
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h"
#define n 6
int main(void){
  hipblasHandle_t handle;
  int j;
  float* x;
  float* y;
  hipMallocManaged(&x, n*sizeof(float));
  hipMallocManaged(&y, n*sizeof(float));
  for(j = 0; j < n; j++){
    x[j] = (float)j;
    y[j] = (float)j;
  }
  
  // device
  float result;
  hipblasCreate(&handle);
  hipblasSdot(handle, n, x, 1, y, 1, &result);
  hipDeviceSynchronize();
  printf("dot product:");
  printf("%7.0f\n", result);
  hipFree(x);
  hipFree(y);
  hipblasDestroy(handle);
  return EXIT_SUCCESS;
}
