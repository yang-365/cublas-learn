
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h"
#define n 6
int main(void){
  hipblasHandle_t handle;  // CUBLAS context
  int j;  // index of the elements
  float *x;
  hipMallocManaged(&x, n*sizeof(float));  // unified mem for x
  for(j = 0; j < n; j++){
    x[j] = (float)j;  // x = {0, 1, 2, 3, 4, 5}
  }
  printf("x: ");
  for(j = 0; j < n; j++){
    printf("%4.0f ", x[j]);
  }
  printf("\n");
  // device 
  hipblasCreate(&handle);
  int result;
  // find smallset absolute value index
  hipblasIsamin(handle, n, x, 1, &result);
  hipDeviceSynchronize();
  printf("mim |x[i]|:%4.0f\n", fabs(x[result-1]));
  // find maximal absolute value index
  hipblasIsamax(handle, n, x, 1, &result);
  printf("max |x[i]|:%4.0f\n", fabs(x[result-1]));
  hipDeviceSynchronize();
  
  hipFree(x);
  hipblasDestroy(handle);
  return EXIT_SUCCESS;

}
