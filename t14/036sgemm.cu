
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h"
#define IDX2C(i,j,ld) (((j)*(ld))+(i))
#define m 6
#define n 4
#define k 5
int main(void){
  hipblasHandle_t handle;
  int i,j;
  float* a;
  float* b;
  float* c;
  
  hipMallocManaged(&a,m*k*sizeof(float));
  hipMallocManaged(&b,k*n*sizeof(float));
  hipMallocManaged(&c,m*n*sizeof(float));

  int ind = 11;
  for(j = 0; j < k; j++){
    for(i = 0; i < m; i++){
      a[IDX2C(i,j,m)] = (float)ind++;
    }
  }

  ind = 11;
  for(j = 0; j < n; j++){
    for(i = 0; i < k; i++){
      b[IDX2C(i,j,k)] = (float)ind++;
    }
  }
  
  ind = 11;
  for(j = 0; j < n; j++){
    for(i = 0; i < m; i++){
      c[IDX2C(i,j,m)] = (float)ind++;
    }
  }
  
  hipblasCreate(&handle);
  float al = 1.0f;
  float bet = 1.0f;
  hipblasSgemm(handle,HIPBLAS_OP_N,HIPBLAS_OP_N,m,n,k,&al,a,m,b,k,&bet,c,m);
  hipDeviceSynchronize();
  for(i = 0; i < m; i++){
    for(j = 0; j < n; j++){
      printf("%7.0f ",c[IDX2C(i,j,m)]);
    }
    printf("\n");
  }
  hipFree(a);
  hipFree(b);
  hipFree(c);
  hipblasDestroy(handle);
  return EXIT_SUCCESS;
}
