
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h"
#define n 6
int main(void){
  hipblasHandle_t handle;
  int j;
  float* x;
  float* y;
  hipMallocManaged(&x, n*sizeof(float));
  hipMallocManaged(&y, n*sizeof(float));
  for(j = 0; j < n; j++){
    x[j] = (float)j;
    y[j] = (float)j*j;
  }

  // device
  float c = 0.5;
  float s = 0.8669254;
  hipblasCreate(&handle);
  hipblasSrot(handle, n, x, 1, y, 1, &c, &s);
  hipDeviceSynchronize();
  printf("x after srot:");
  for(j = 0; j < n; j++){
    printf("%7.2f ", x[j]);
  }
  printf("\n");

  printf("y after srot:");
  for(j = 0; j < n; j++){
    printf("%7.2f ", y[j]);
  }
  printf("\n");
  hipFree(x);
  hipFree(y);
  hipblasDestroy(handle);
  return EXIT_SUCCESS;

}
