
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "hipblas.h"
#define n 6
int main(void){
  hipblasHandle_t handle;  // CUBLAS context
  int j;
  float* x;
  float* y;
  hipMallocManaged(&x, n*sizeof(float));
  for(j  = 0; j < n; j++){
    x[j] = (float)j;
  }
  hipMallocManaged(&y, n*sizeof(float));
  for(j = 0; j < n; j++){
    y[j] = (float)j;
  }
  printf("x:");
  for(j = 0; j < n; j++){
    printf("%4.0f ", x[j]);
  }
  printf("\n");
  printf("y:");
  for(j = 0; j < n; j++){
    printf("%4.0f ", y[j]);
  }
  printf("\n");

  // device
  hipblasCreate(&handle);
  float al = 2.0;
  hipblasSaxpy(handle, n, &al, x, 1, y, 1);
  hipDeviceSynchronize();

  printf("y after Saxpy:");
  for(j = 0; j < n; j++){
    printf("%4.0f ", y[j]);
  }
  printf("\n");

  hipFree(x);
  hipFree(y);
  return EXIT_SUCCESS;
}
